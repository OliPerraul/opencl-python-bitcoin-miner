#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include "sha256/sha256.cuh"

#define PrevHashSize 32U
#define MerkleHashSize 32U
#define BlockSize 80U


typedef uint32_t uint;

__device__
// size = sizeof(start) < sizeof(sequence); 
bool prefix_0(char* sequence, uint difficulty)
{
    for(uint i = 0; i < difficulty; i++)
    {
        if(sequence[i] != '\0')
            return false;
    }

    return true;
}

__device__
void hash_sha256(char *content, char *hash){

	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, reinterpret_cast<BYTE*>(content), BlockSize);
    sha256_final(&ctx, reinterpret_cast<BYTE*>(hash));
    
    sha256_init(&ctx);
	sha256_update(&ctx, reinterpret_cast<BYTE*>(hash), BlockSize);
    sha256_final(&ctx, reinterpret_cast<BYTE*>(hash));
    
    sha256_init(&ctx);
	sha256_update(&ctx, reinterpret_cast<BYTE*>(hash), BlockSize);
	sha256_final(&ctx, reinterpret_cast<BYTE*>(hash));
}

__global__ void compute_blockchain(
    uint numNonceTrials,
    uint version,
    // Timestamp greater than median timestamp over of last 11 blocks
    uint timestamp,
    uint difficulty,
    const char* prevHash,
    const char* merkleHash,
    bool* found,
    char* outputBuffer,
    char* blockOutputBuffer)
{
    const uint threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    const uint threadNumInBlock = threadIdx.x + threadIdx.y *(blockDim.x) + threadIdx.z * (blockDim.x*blockDim.y);
    const uint blockNumInGrid = blockIdx.x  + blockIdx.y * gridDim.x + blockIdx.z * ( gridDim.x * gridDim.y);
    
    // nonce = threadId * numNonceTrials
    uint nonce = (blockNumInGrid * threadsPerBlock + threadNumInBlock) * numNonceTrials;
    //

    char block_content[BlockSize];
    char check_buffer[BlockSize];

    // res = input->version + self->prevHash + self->merkleHash + self->time + self->difficulty + self->nonce
    memcpy(&(block_content[0]), &version, 4);
    memcpy(&(block_content[4]), &prevHash, PrevHashSize);
    memcpy(&(block_content[PrevHashSize + 4]), &merkleHash, MerkleHashSize);
    memcpy(&(block_content[PrevHashSize + MerkleHashSize + 4]), &timestamp, 4);
    memcpy(&(block_content[PrevHashSize + MerkleHashSize + 2*4]), &difficulty, 4);

    uint i = 0;
    do
    {
        if(i >= numNonceTrials)
        {
            break;
        }

        memcpy(&(block_content[PrevHashSize + MerkleHashSize + 3*4]), &nonce, 4);
        hash_sha256(block_content, check_buffer);
        
        if(prefix_0(check_buffer, difficulty))
        {
            (*found) = true;
            memcpy(&(outputBuffer[0]), &(check_buffer[0]), BlockSize);
            memcpy(&(blockOutputBuffer[0]), &(block_content[0]), BlockSize);
        }

        i++;
        nonce++;

    } while(!(*found));
}
